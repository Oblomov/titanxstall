#include "hip/hip_runtime.h"
/// Copyright (C) 2016 Giuseppe Bilotta <giuseppe.bilotta@gmail.com>
/// License: GPLv3

#include <sstream>
#include <fstream>
#include <stdexcept>

#include <unistd.h>
#include <signal.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <ctime>

#include <hip/hip_runtime.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/system/cuda/execution_policy.h>

// declare here for use by cached_alloc.h, should be refactored
void check(const char *file, unsigned long line, const char *func);
#define CHECK(func) check(__FILE__, __LINE__, func)

#define restrict __restrict__

typedef unsigned int uint;
typedef unsigned int hashKey;
typedef ushort4 particleinfo;

enum ParticleType {
	PT_FLUID = 0,
	PT_BOUNDARY,
	PT_VERTEX,
	PT_TESTPOINT,
	PT_NONE
};

static __forceinline__ __host__ __device__ __attribute__((pure)) const ushort& type(const particleinfo &info)
{ return info.x; }

static __forceinline__ __host__ __device__ __attribute__((pure)) uint id(const particleinfo &info)
{ return (uint)(info.z) | ((uint)(info.w) << 16); }

#define PART_FLAG_SHIFT	3
#define PART_TYPE_MASK	((1<<PART_FLAG_SHIFT)-1)
#define PART_TYPE(f) ParticleType(type(f) & PART_TYPE_MASK)

// some defines to make the compare functor and thrust sort key invokation more legible
typedef thrust::tuple<hashKey, particleinfo> hash_info_pair;

typedef thrust::device_ptr<particleinfo> thrust_info_ptr;
typedef thrust::device_ptr<hashKey> thrust_hash_ptr;
typedef thrust::device_ptr<uint> thrust_uint_ptr;

typedef thrust::tuple<thrust_hash_ptr, thrust_info_ptr> hash_info_iterator_pair;
typedef thrust::zip_iterator<hash_info_iterator_pair> key_iterator;

enum SPHFormulation { SPH_F1 = 1, };

enum BoundaryType { LJ_BOUNDARY, };

enum Periodicity { PERIODIC_NONE = 0, };

typedef uint64_t flag_t;

#define ENABLE_NONE ((flag_t)0)

enum RheologyType { NEWTONIAN, };

enum TurbulenceModel { LAMINAR_FLOW, KEPSILON, };

enum ComputationalViscosityType { KINEMATIC, };

enum ViscousModel { MORRIS, };

enum AverageOperator { ARITHMETIC, };

template<
	RheologyType _rheologytype = NEWTONIAN,
	TurbulenceModel _turbmodel = LAMINAR_FLOW,
	ComputationalViscosityType _compvisc = KINEMATIC,
	ViscousModel _viscmodel = MORRIS,
	AverageOperator _avgop = ARITHMETIC,
	flag_t _simflags = ENABLE_NONE,
	// is this a constant-viscosity formulation?
	bool _is_const_visc = (
		(_simflags != ENABLE_NONE) &&
		(_rheologytype == NEWTONIAN) &&
		(_turbmodel != KEPSILON)
	)
>
struct FullViscSpec {
	static constexpr RheologyType rheologytype = _rheologytype;
	static constexpr TurbulenceModel turbmodel = _turbmodel;
	static constexpr ComputationalViscosityType compvisc = _compvisc;
	static constexpr ViscousModel viscmodel = _viscmodel;
	static constexpr AverageOperator avgop = _avgop;
	static constexpr flag_t simflags = _simflags;

	static constexpr bool is_const_visc = _is_const_visc;
};

class AbstractEngine
{
public:
	virtual void sort(particleinfo *info, hashKey *hash, uint *partidx, uint numParticles) = 0;
};

template<SPHFormulation sph_formulation, typename ViscSpec, BoundaryType boundarytype, Periodicity periodicbound, flag_t simflags,
	bool neibcount>
class CUDAEngine : public AbstractEngine
{
public:

/// Functor to sort particles by hash (cell), and
/// by fluid number within the cell
struct ptype_hash_compare :
	public thrust::binary_function<hash_info_pair, hash_info_pair, bool>
{
	typedef thrust::tuple<hashKey, particleinfo> value_type;

	__host__ __device__
	bool operator()(const value_type& a, const value_type& b)
	{
		uint	ha(thrust::get<0>(a)),
			hb(thrust::get<0>(b));
		particleinfo	pa(thrust::get<1>(a)),
				pb(thrust::get<1>(b));

		if (ha == hb) {
			const ParticleType pta = PART_TYPE(pa),
				ptb = PART_TYPE(pb);
			if (pta == ptb)
				return id(pa) < id(pb);
			return (pta < ptb);
		}
		return (ha < hb);
	}
};

void
sort(particleinfo *info, hashKey *hash, uint *partidx, uint numParticles)
{
	thrust_info_ptr particleInfo =
		thrust::device_pointer_cast(info);
	thrust_hash_ptr particleHash =
		thrust::device_pointer_cast(hash);
	thrust_uint_ptr particleIndex =
		thrust::device_pointer_cast(partidx);

	for (uint i = 0; i < numParticles; ++i) {
		printf("BEFORE: %d: %d %d %d %d %d %d\n", i, info[i].x, info[i].y, info[i].z, info[i].w,
			hash[i], partidx[i]);
	}

	ptype_hash_compare comp;

	key_iterator key_start(thrust::make_tuple(particleHash, particleInfo));
	key_iterator key_end(thrust::make_tuple(
			particleHash + numParticles,
			particleInfo + numParticles));

	if (numParticles > 0)
		thrust::sort_by_key(key_start, key_end, particleIndex, comp);

	for (uint i = 0; i < numParticles; ++i) {
		printf("AFTER: %d: %d %d %d %d %d %d\n", i, info[i].x, info[i].y, info[i].z, info[i].w,
			hash[i], partidx[i]);
	}
}

};

__global__ void
initIdx(uint* partidx, uint numParticles)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	partidx[index] = index;
}

using namespace std;

void check(const char *file, unsigned long line, const char *func)
{
	hipError_t err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		stringstream errmsg;
		errmsg << file << ":" << line << " in " << func
			<< ": runtime API error " << err << " (" << hipGetErrorString(err) << ")";
		throw runtime_error(errmsg.str());
	}
}


int main(int argc, char *argv[])
{
	uint numParticles = 84600;
	uint device = 0;

	const char * const * arg = argv + 1;
	while (argc > 1) {
		if (!strcmp(*arg, "--device")) {
			if (argc < 2)
				throw invalid_argument("please specify a device");
			--argc;
			++arg;
			device = atoi(*arg);
		}
		--argc;
		++arg;

	}

	hipSetDevice(device);
	CHECK("set device");

	particleinfo *h_info = NULL, *d_info = NULL;
	hashKey *h_hash = NULL, *d_hash = NULL;
	uint *h_partidx = NULL, *d_partidx = NULL;

	float4 *d_pos = NULL, *d_vel = NULL, *d_forces = NULL;

	uint *d_cellStart = NULL, *d_cellEnd = NULL;
	ushort *d_neibsList = NULL;

	uint numCells = 10455;
	uint neibsListSize = numParticles*128;


	h_info = new particleinfo[numParticles];
	h_hash = new hashKey[numParticles];
	h_partidx = new uint[numParticles];

	hipMallocManaged(&d_pos, numParticles*sizeof(*d_pos));
	hipMallocManaged(&d_vel, numParticles*sizeof(*d_vel));
	hipMallocManaged(&d_info, numParticles*sizeof(*d_info));

	hipMallocManaged(&d_forces, numParticles*sizeof(*d_forces));
	hipMemset(d_forces, 0, numParticles*sizeof(*d_forces));

	hipMallocManaged(&d_cellStart, numCells*sizeof(*d_cellStart));
	hipMemset(d_cellStart, -1, numCells*sizeof(*d_cellStart));
	hipMallocManaged(&d_cellEnd, numCells*sizeof(*d_cellEnd));
	hipMemset(d_cellEnd, -1, numCells*sizeof(*d_cellEnd));

	hipMallocManaged(&d_hash, numParticles*sizeof(*d_hash));
	hipMallocManaged(&d_partidx, numParticles*sizeof(*d_partidx));

	hipMallocManaged(&d_neibsList, neibsListSize*sizeof(*d_neibsList));
	hipMemset(d_neibsList, -1,  neibsListSize*sizeof(*d_neibsList));

	ifstream data("data.idx");
	for (uint i = 0; i < numParticles; ++i) {
		particleinfo pi;
		data >> pi.x >> pi.y >> pi.z >> pi.w;

		h_info[i] = pi;

		data >> h_hash[i] >> h_partidx[i];
	}

	hipMemcpy(d_info, h_info, numParticles*sizeof(*h_info), hipMemcpyHostToDevice);
	hipMemcpy(d_hash, h_hash, numParticles*sizeof(*h_hash), hipMemcpyHostToDevice);

	// On GPUSPH, partidx is actually initialized on device
	initIdx<<<(numParticles + 256 - 1)/256, 256>>>(d_partidx, numParticles);
	CHECK("initIdx");

	using MyViscSpec = FullViscSpec<>;

	AbstractEngine *engine = new CUDAEngine<SPH_F1, MyViscSpec, LJ_BOUNDARY, PERIODIC_NONE, ENABLE_NONE, true>();

	engine->sort(d_info, d_hash, d_partidx, numParticles);

	hipFree(d_partidx);
	hipFree(d_hash);
	hipFree(d_info);

	delete[] h_partidx;
	delete[] h_hash;
	delete[] h_info;
}
